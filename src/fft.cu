#include "hip/hip_runtime.h"
/*
 * =====================================================================================
 *
 *       Filename: fft.cu
 *
 *    Description:  
 *
 *        Version:  1.0
 *        Created:  13.2.2013 22:59:33
 *       Compiler:  gcc
 *
 *         Author:  Robert David
 *
 * =====================================================================================
 */
#include <stdlib.h>
#include <stdio.h>
#include <hipfft/hipfft.h>

#include "poly.h"

#define C_SIZE (2*poly_size[0])
#define C_SIZE_COMPLEX (C_SIZE*sizeof(hipfftComplex))

#define THREADS 1024

/* init and print GPU */
void cuda_init( int devID )
{
	hipDeviceProp_t deviceProp;

	hipGetDeviceProperties( &deviceProp, devID );
	if( deviceProp.major < 1 )
	{
		fprintf( stderr, "gpuDeviceInit(): GPU device does not support CUDA.\n" );
		exit( -1 );
	}
	hipSetDevice(devID);
	fprintf( stderr, "gpuDeviceInit() CUDA Device [%d]: \"%s\n", devID, deviceProp.name );
}

/* just multiply complex numbers */
__global__ void multiply_points( hipfftComplex * dev_a, hipfftComplex * dev_b, hipfftComplex * dev_c, ul_int size )
{
	ul_int i = (blockIdx.x)*blockDim.x + (threadIdx.x);

	if(i<size)
	{
		dev_c[i].x = dev_a[i].x * dev_b[i].x - dev_a[i].y * dev_b[i].y;
		dev_c[i].y = dev_a[i].x * dev_b[i].y + dev_a[i].y * dev_b[i].x;
	}
}

/* retype and copy poly to complex host_x and copy to GPU */
void copy_mem_dev( hipfftComplex * host_a, hipfftComplex * host_b, hipfftComplex * dev_a, hipfftComplex * dev_b )
{
	ul_int i;
	for( i=0 ; i<poly_size[0] ; i++ )
	{
		host_a[i].x = (float)poly[A][i];
		host_b[i].x = (float)poly[B][i];
	}

	hipMemcpy( dev_a, host_a, C_SIZE_COMPLEX, hipMemcpyHostToDevice );
	hipMemcpy( dev_b, host_b, C_SIZE_COMPLEX, hipMemcpyHostToDevice );
}

/* copy array C from GPU and retype real part to int */
void copy_mem_host( hipfftComplex * host_c, hipfftComplex * dev_c )
{
	hipMemcpy( host_c, dev_c, C_SIZE_COMPLEX, hipMemcpyDeviceToHost );

	ul_int i;
	for( i=0 ; i<C_SIZE ; i++ )
	{
		poly[C][i] = (unsigned int)(host_c[i].x/C_SIZE);
	}
}

extern "C" void calculate_cuda( void )
{
	hipfftHandle plan;
	hipfftComplex * cu_poly_a;
	hipfftComplex * cu_poly_b;
	hipfftComplex * cu_poly_c;
	hipfftComplex * cu_host_poly_a;
	hipfftComplex * cu_host_poly_b;
	hipfftComplex * cu_host_poly_c;
	ul_int blocks = C_SIZE/THREADS + 1;

	cuda_init( 0 );

	/* alloc required arrays */
	hipMalloc( (void**)&cu_poly_a, C_SIZE_COMPLEX );
	hipMalloc( (void**)&cu_poly_b, C_SIZE_COMPLEX );
	hipMalloc( (void**)&cu_poly_c, C_SIZE_COMPLEX );
	cu_host_poly_a = (hipfftComplex*) calloc( sizeof(hipfftComplex), C_SIZE );
	cu_host_poly_b = (hipfftComplex*) calloc( sizeof(hipfftComplex), C_SIZE );
	cu_host_poly_c = (hipfftComplex*) calloc( sizeof(hipfftComplex), C_SIZE );

	/* copy data to GPU */
	copy_mem_dev( cu_host_poly_a, cu_host_poly_b, cu_poly_a, cu_poly_b );

	/* set up cufft */
	hipfftPlan1d( &plan, C_SIZE, HIPFFT_C2C, 1 );

	/* call fft transformation on each input array */
	hipfftExecC2C( plan, cu_poly_a, cu_poly_a, HIPFFT_FORWARD );
	hipfftExecC2C( plan, cu_poly_b, cu_poly_b, HIPFFT_FORWARD );

	/* multiply complex products of fft transformation */
	multiply_points<<<blocks,THREADS>>>( cu_poly_a, cu_poly_b, cu_poly_c, C_SIZE );

	/* call inverse fft transformation on output array */
	hipfftExecC2C( plan, cu_poly_c, cu_poly_c, HIPFFT_BACKWARD );

	/* copy data from GPU */
	copy_mem_host( cu_host_poly_c, cu_poly_c );

	/* free memory */
	hipfftDestroy( plan );
	hipFree( cu_poly_a );
	hipFree( cu_poly_b );
	hipFree( cu_poly_c );
	free( cu_host_poly_a );
	free( cu_host_poly_b );
	free( cu_host_poly_c );
}
