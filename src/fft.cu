#include "hip/hip_runtime.h"
/*
 * =====================================================================================
 *
 *       Filename: fft.cu
 *
 *    Description:  
 *
 *        Version:  1.0
 *        Created:  13.2.2013 22:59:33
 *       Compiler:  gcc
 *
 *         Author:  Robert David
 *
 * =====================================================================================
 */
#include <stdlib.h>
#include <stdio.h>
#include <hipfft/hipfft.h>
#include <math.h>
#include <complex.h>

#include "poly.h"

#define C_SIZE (2*poly_size[0])
#define THREADS 1024

__global__ void multiply_points( 
		hipfftComplex * dev_a,
		hipfftComplex * dev_b,
		hipfftComplex * dev_c,
		ul_int size )
{
	ul_int i = (blockIdx.x)*blockDim.x + (threadIdx.x);

	if(i<size)
	{
		dev_c[i].x = dev_a[i].x * dev_b[i].x - dev_a[i].y * dev_b[i].y;
		dev_c[i].y = dev_a[i].x * dev_b[i].y + dev_a[i].y * dev_b[i].x;
	}
}

void copy_mem_dev( 
		hipfftComplex * host_a,
		hipfftComplex * host_b,
		hipfftComplex * dev_a,
		hipfftComplex * dev_b )
{
	ul_int i;
	for(i=0 ; i<poly_size[0] ; i++)
	{
		host_a[i].x = (float)poly[A][i];
		host_b[i].x = (float)poly[B][i];
	}

	hipMemcpy(dev_a, host_a, C_SIZE*sizeof(hipfftComplex), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, host_b, C_SIZE*sizeof(hipfftComplex), hipMemcpyHostToDevice);
}

void copy_mem_host( hipfftComplex * host_c, hipfftComplex * dev_c )
{
	hipMemcpy(host_c, dev_c, C_SIZE*sizeof(hipfftComplex), hipMemcpyDeviceToHost);

	ul_int i;
	for(i=0 ; i<C_SIZE ; i++)
	{
		poly[C][i] = (unsigned int)(host_c[i].x/C_SIZE);
	}
}

extern "C" void calculate_cuda( void )
{
	hipfftHandle plan;
	hipfftComplex * cu_poly_a;
	hipfftComplex * cu_poly_b;
	hipfftComplex * cu_poly_c;
	hipfftComplex * cu_host_poly_a;
	hipfftComplex * cu_host_poly_b;
	hipfftComplex * cu_host_poly_c;
	ul_int blocks = C_SIZE/THREADS + 1;

	hipMalloc((void**)&cu_poly_a, sizeof(hipfftComplex)*C_SIZE );
	hipMalloc((void**)&cu_poly_b, sizeof(hipfftComplex)*C_SIZE );
	hipMalloc((void**)&cu_poly_c, sizeof(hipfftComplex)*C_SIZE );
	cu_host_poly_a = (hipfftComplex*) calloc(sizeof(hipfftComplex),C_SIZE );
	cu_host_poly_b = (hipfftComplex*) calloc(sizeof(hipfftComplex),C_SIZE );
	cu_host_poly_c = (hipfftComplex*) calloc(sizeof(hipfftComplex),C_SIZE );

	copy_mem_dev( cu_host_poly_a, cu_host_poly_b, cu_poly_a, cu_poly_b);

	hipfftPlan1d(&plan, C_SIZE, HIPFFT_C2C, 1);

	hipfftExecC2C(plan, cu_poly_a, cu_poly_a, HIPFFT_FORWARD);
	hipfftExecC2C(plan, cu_poly_b, cu_poly_b, HIPFFT_FORWARD);

	multiply_points<<<blocks,THREADS>>>( cu_poly_a, cu_poly_b, cu_poly_c, C_SIZE );

	hipfftExecC2C(plan, cu_poly_c, cu_poly_c, HIPFFT_BACKWARD);

	copy_mem_host( cu_host_poly_c, cu_poly_c );

	hipfftDestroy(plan);
	hipFree(cu_poly_a);
	hipFree(cu_poly_b);
	hipFree(cu_poly_c);
	free(cu_host_poly_a);
	free(cu_host_poly_b);
	free(cu_host_poly_c);
}
