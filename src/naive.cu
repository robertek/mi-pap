#include "hip/hip_runtime.h"
/*
 * =====================================================================================
 *
 *       Filename:  naive.cu
 *
 *    Description:  
 *
 *        Version:  1.0
 *        Created:  13.2.2013 22:59:33
 *       Compiler:  gcc
 *
 *         Author:  Robert David
 *
 * =====================================================================================
 */
#include <stdlib.h>

#include "poly.h"

__global__ void multiply( unsigned long int * data_a, 
		unsigned long int * data_b, unsigned long int * data_c, unsigned long int size )
{
	int i,j;
	unsigned long int tmp;

	i = threadIdx.x;
	j = blockIdx.x;

	data_c[i+j] = data_a[i]*data_b[j];
	//tmp = data_a[i]*data_b[j];
	//atomicAdd( &data_c[i+j], tmp );
}

extern "C" void calculate_cuda( void )
{
	unsigned long int * data_a;
	unsigned long int * data_b;
	unsigned long int * data_c;
	hipMalloc( &data_a, sizeof(long int)*poly_size[A] );
	hipMalloc( &data_b, sizeof(long int)*poly_size[A] );
	hipMalloc( &data_c, 2*sizeof(long int)*poly_size[A] );

	hipMemcpy( data_a, poly[A], sizeof(long int)*poly_size[A], hipMemcpyHostToDevice );
	hipMemcpy( data_b, poly[B], sizeof(long int)*poly_size[A], hipMemcpyHostToDevice );

	multiply<<<poly_size[A],poly_size[A]>>>( data_a, data_b, data_c, poly_size[A] );

	hipMemcpy( poly[C], data_c, 2*sizeof(long int)*poly_size[A], hipMemcpyDeviceToHost );

	hipFree( data_a );
	hipFree( data_b );
	hipFree( data_c );
}
