#include "hip/hip_runtime.h"
/*
 * =====================================================================================
 *
 *       Filename:  naive.cu
 *
 *    Description:  
 *
 *        Version:  1.0
 *        Created:  13.2.2013 22:59:33
 *       Compiler:  gcc
 *
 *         Author:  Robert David
 *
 * =====================================================================================
 */
#include <stdlib.h>

#include "poly.h"

__global__ void multiply( int * data_a, int * data_b, int * data_c, int size )
{
	int i,j;
	int tmp;

	i = threadIdx.x;
	j = blockIdx.x;

	tmp = data_a[i]*data_b[j];
	atomicAdd( &data_c[i+j], tmp );
}

extern "C" void calculate_cuda( void )
{
	int * data_a;
	int * data_b;
	int * data_c;
	hipMalloc( &data_a, sizeof(int)*poly_size[A] );
	hipMalloc( &data_b, sizeof(int)*poly_size[A] );
	hipMalloc( &data_c, 2*sizeof(int)*poly_size[A] );

	hipMemcpy( data_a, poly[A], sizeof(int)*poly_size[A], hipMemcpyHostToDevice );
	hipMemcpy( data_b, poly[B], sizeof(int)*poly_size[A], hipMemcpyHostToDevice );

	multiply<<<poly_size[A],poly_size[A]>>>( data_a, data_b, data_c, poly_size[A] );

	hipMemcpy( poly[C], data_c, 2*sizeof(int)*poly_size[A], hipMemcpyDeviceToHost );

	hipFree( data_a );
	hipFree( data_b );
	hipFree( data_c );
}
