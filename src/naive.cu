#include "hip/hip_runtime.h"
/*
 * =====================================================================================
 *
 *       Filename:  naive.cu
 *
 *    Description:  
 *
 *        Version:  1.0
 *        Created:  13.2.2013 22:59:33
 *       Compiler:  gcc
 *
 *         Author:  Robert David
 *
 * =====================================================================================
 */
#include <stdlib.h>

#include "poly.h"

__global__ void multiply( unsigned long int * data_a, 
		unsigned long int * data_b, unsigned long int * data_c, unsigned long int size )
{
	int i,j;
	unsigned long int tmp;

	for( i=threadIdx.x*size ; i<(threadIdx.x+1)*size ; i++ )
		for( j=blockIdx.x*size ; j<(blockIdx.x+1)*size ; j++ )
			data_c[i+j] = data_a[i]*data_b[j];
}

extern "C" void calculate_cuda( void )
{
	unsigned long int * data_a;
	unsigned long int * data_b;
	unsigned long int * data_c;
	hipMalloc( &data_a, sizeof(long int)*poly_size[A] );
	hipMalloc( &data_b, sizeof(long int)*poly_size[A] );
	hipMalloc( &data_c, 2*sizeof(long int)*poly_size[A] );

	hipMemcpy( data_a, poly[A], sizeof(long int)*poly_size[A], hipMemcpyHostToDevice );
	hipMemcpy( data_b, poly[B], sizeof(long int)*poly_size[A], hipMemcpyHostToDevice );

	multiply<<<32,32>>>( data_a, data_b, data_c, poly_size[A]/32 );

	hipMemcpy( poly[C], data_c, 2*sizeof(long int)*poly_size[A], hipMemcpyDeviceToHost );

	hipFree( data_a );
	hipFree( data_b );
	hipFree( data_c );
}
